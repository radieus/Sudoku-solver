#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>  
#include <string.h>
#include <time.h>
#include <math.h>
#include <inttypes.h>

#include "CudaSudoku.cu"
#include "samples.h"



#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}



int main(int argc, char* argv[]) {

#pragma region Delclaration
    hipEvent_t event1,event2;
    
    uint64_t test[N];
    uint64_t check[N];
    uint64_t fun[N];

    uint64_t *test64_1;
    uint64_t *test64_2;
    int *board_index;

    const int sk = pow(2,27);
    int host_count;
    int threadsPerBlock = 256;
    int maxBlocks;
    int zeros;
    params_t params;
    float dt_ms;

    gpuErrchk(hipMalloc(&test64_1,sk*sizeof(uint64_t)));
    gpuErrchk(hipMalloc(&test64_2,sk*sizeof(uint64_t)));
    gpuErrchk(hipMalloc(&board_index,sizeof(int)));

    memset(test,0,N*sizeof(uint64_t));
    memset(check,0,N*sizeof(uint64_t));
    memset(fun,0,N*sizeof(uint64_t));
    gpuErrchk(hipMemset(board_index,0,sizeof(int)));
    gpuErrchk(hipMemset(test64_1,0,sk*sizeof(uint64_t)));
    gpuErrchk(hipMemset(test64_2,0,sk*sizeof(uint64_t)));



    gpuErrchk(hipEventCreate(&event1));
    gpuErrchk(hipEventCreate(&event2));


    setup_board(test,test9);
    
    printBoard(test);

    zeros=count_zeros(test);
    gpuErrchk(hipMemcpy(test64_1,test,N*sizeof(uint64_t),hipMemcpyHostToDevice));

    gpuErrchk(hipEventRecord(event1));

    params=find_epmty_index(test,0,0);

    printf("Empty index %i : %i\n",params.row, params.col);

    cudaBFSSudoku<<<1,N>>>(test64_1, test64_2, 1, board_index,params.row,params.col);

    gpuErrchk(hipMemcpy(&fun, test64_2, N*sizeof(uint64_t), hipMemcpyDeviceToHost))
    params=find_epmty_index(fun,params.row,params.col);


    
    for (int i = 0; i<zeros; i++) {

        gpuErrchk(hipMemcpy(&host_count, board_index, sizeof(int), hipMemcpyDeviceToHost));

        printf("total boards after an iteration %d: %d\n", i, host_count);

        gpuErrchk(hipMemset(board_index, 0, sizeof(int)));

        maxBlocks=(N*host_count+threadsPerBlock-1)/threadsPerBlock;

        if (i % 2 == 0) {
            cudaBFSSudoku<<<maxBlocks,threadsPerBlock>>>(test64_2, test64_1, host_count, board_index,params.row,params.col);
            gpuErrchk(hipMemcpy(&fun, test64_1, N*sizeof(uint64_t), hipMemcpyDeviceToHost));
            params=find_epmty_index(fun,params.row,params.col);
        }
        else {
            cudaBFSSudoku<<<maxBlocks,threadsPerBlock>>>(test64_1, test64_2, host_count, board_index,params.row,params.col);
            gpuErrchk(hipMemcpy(&fun, test64_2, N*sizeof(uint64_t), hipMemcpyDeviceToHost));
            params=find_epmty_index(fun,params.row,params.col);
        }
    }

    gpuErrchk(hipMemcpy(&host_count, board_index, sizeof(int), hipMemcpyDeviceToHost));
    
    if(zeros % 2 == 0){
        gpuErrchk(hipMemcpy(&check, test64_1, N*sizeof(uint64_t), hipMemcpyDeviceToHost));
    }
    else{
        gpuErrchk(hipMemcpy(&check, test64_2, N*sizeof(uint64_t), hipMemcpyDeviceToHost));
    }
    
    printf("new number of boards retrieved is %d\n", host_count);
    printBoard(check);


    gpuErrchk(hipEventRecord(event2));
    gpuErrchk(hipEventSynchronize(event2));
    gpuErrchk(hipDeviceSynchronize());

    hipEventElapsedTime(&dt_ms, event1,event2);
    printf("Time : %f",dt_ms);

    gpuErrchk(hipFree(test64_1));
    gpuErrchk(hipFree(test64_2));
    gpuErrchk(hipFree(board_index));

    return 0; 
}

