#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>  
#include <string.h>
#include <time.h>
#include <math.h>
#include <inttypes.h>

#include "CudaSudoku.cu"
#include "samples.h"



#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}



int main(int argc, char* argv[]) {

#pragma region Delclaration
    hipEvent_t event1,event2;
    
    int test[N];
    int check[N];
    int fun[N];

    int *test64_1;
    int *test64_2;
    int *board_index;

    const int sk = pow(2,27);
    int host_count;
    int threadsPerBlock = 256;
    int maxBlocks;
    int zeros;
    params_t params;
    float dt_ms;

    gpuErrchk(hipMalloc(&test64_1,sk*sizeof(int)));
    gpuErrchk(hipMalloc(&test64_2,sk*sizeof(int)));
    gpuErrchk(hipMalloc(&board_index,sizeof(int)));

    memset(test,0,N*sizeof(int));
    memset(check,0,N*sizeof(int));
    memset(fun,0,N*sizeof(int));
    gpuErrchk(hipMemset(board_index,0,sizeof(int)));
    gpuErrchk(hipMemset(test64_1,0,sk*sizeof(int)));
    gpuErrchk(hipMemset(test64_2,0,sk*sizeof(int)));



    gpuErrchk(hipEventCreate(&event1));
    gpuErrchk(hipEventCreate(&event2));


    setup_board(test,test9);
    
    printBoard(test);

    zeros=count_zeros(test);
    gpuErrchk(hipMemcpy(test64_1,test,N*sizeof(int),hipMemcpyHostToDevice));

    gpuErrchk(hipEventRecord(event1));

    params=find_epmty_index(test,0,0);

    printf("Empty index %i : %i\n",params.row, params.col);

    cudaBFSSudoku<<<1,N>>>(test64_1, test64_2, 1, board_index,params.row,params.col);

    gpuErrchk(hipMemcpy(&fun, test64_2, N*sizeof(int), hipMemcpyDeviceToHost))
    params=find_epmty_index(fun,params.row,params.col);


    
    for (int i = 0; i<zeros; i++) {

        gpuErrchk(hipMemcpy(&host_count, board_index, sizeof(int), hipMemcpyDeviceToHost));

        printf("total boards after an iteration %d: %d\n", i, host_count);

        gpuErrchk(hipMemset(board_index, 0, sizeof(int)));

        maxBlocks=(N*host_count+threadsPerBlock-1)/threadsPerBlock;

        if (i % 2 == 0) {
            cudaBFSSudoku<<<maxBlocks,threadsPerBlock>>>(test64_2, test64_1, host_count, board_index,params.row,params.col);
            gpuErrchk(hipMemcpy(&fun, test64_1, N*sizeof(int), hipMemcpyDeviceToHost));
            params=find_epmty_index(fun,params.row,params.col);
        }
        else {
            cudaBFSSudoku<<<maxBlocks,threadsPerBlock>>>(test64_1, test64_2, host_count, board_index,params.row,params.col);
            gpuErrchk(hipMemcpy(&fun, test64_2, N*sizeof(int), hipMemcpyDeviceToHost));
            params=find_epmty_index(fun,params.row,params.col);
        }
    }

    gpuErrchk(hipMemcpy(&host_count, board_index, sizeof(int), hipMemcpyDeviceToHost));
    
    if(zeros % 2 == 0){
        gpuErrchk(hipMemcpy(&check, test64_1, N*sizeof(int), hipMemcpyDeviceToHost));
    }
    else{
        gpuErrchk(hipMemcpy(&check, test64_2, N*sizeof(int), hipMemcpyDeviceToHost));
    }
    
    printf("new number of boards retrieved is %d\n", host_count);
    printBoard(check);


    gpuErrchk(hipEventRecord(event2));
    gpuErrchk(hipEventSynchronize(event2));
    gpuErrchk(hipDeviceSynchronize());

    hipEventElapsedTime(&dt_ms, event1,event2);
    printf("Time : %f",dt_ms);

    gpuErrchk(hipFree(test64_1));
    gpuErrchk(hipFree(test64_2));
    gpuErrchk(hipFree(board_index));

    return 0; 
}

