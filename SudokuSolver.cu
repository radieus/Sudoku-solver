#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>  
#include <string.h>
#include <time.h>
#include <math.h>
#include <inttypes.h>

#include "CudaSudoku.cu"
#include "samples.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void printBoard(int *board) {
    for (int i = 0; i < N; i++) {
        if (i % n == 0) {
            printf("-----------------------\n");
        }

        for (int j = 0; j < N; j++) {
            if (j % n == 0) {
            printf("| ");
            }
            printf("%d ", board[i * N + j]);
        }

        printf("|\n");
    }
    printf("-----------------------\n");
}


int main(int argc, char* argv[]) {

#pragma region Delclaration
    hipEvent_t event1,event2;
    
    uint64_t test[N];
    uint64_t check[N];
    uint64_t fun[N];

    uint64_t *new_boards;
    uint64_t *old_boards;
    int *board_index;


    const int sk = pow(2,27);
    int host_count;
    int threadsPerBlock = 256;
    int maxBlocks;
    int zeros;
    params_t params;
    float dt_ms;

    gpuErrchk(hipMallocManaged(&new_boards,sk*sizeof(uint64_t)));
    gpuErrchk(hipMallocManaged(&old_boards,sk*sizeof(uint64_t)));
    gpuErrchk(hipMallocManaged(&board_index,sizeof(int)));

    memset(test,0,N*sizeof(uint64_t));
    memset(check,0,N*sizeof(uint64_t));
    memset(fun,0,N*sizeof(uint64_t));

    board_index = 0;
    new_boards = 0;
    old_boards = 0;

    gpuErrchk(hipEventCreate(&event1));
    gpuErrchk(hipEventCreate(&event2));

    //------------------------------------------------------------------------------------------------------------------------
    setup_board(new_boards,test9);
    //load("sudoku.txt", test);
    //------------------------------------------------------------------------------------------------------------------------
    
    print_sudoku_from_b64(new_boards);

    zeros=count_zeros(new_boards);
    //gpuErrchk(hipMemcpy(new_boards,test,N*sizeof(uint64_t),hipMemcpyHostToDevice));

    gpuErrchk(hipEventRecord(event1));

    params=find_epmty_index(new_boards,0,0);

    printf("Empty index %i : %i\n", params.row, params.col);

    cudaBFSSudoku<<<1,N>>>(new_boards, old_boards, 1, board_index, params.row, params.col);

    //gpuErrchk(hipMemcpy(&fun, old_boards, N*sizeof(uint64_t), hipMemcpyDeviceToHost))
    params=find_epmty_index(old_boards, params.row, params.col);


    
    for (int i = 0; i<zeros; i++) {

        //gpuErrchk(hipMemcpy(&host_count, board_index, sizeof(int), hipMemcpyDeviceToHost));

        printf("total boards after an iteration %d: %d\n", i, host_count);

        gpuErrchk(hipMemset(board_index, 0, sizeof(int)));

        maxBlocks=(N*host_count+threadsPerBlock-1)/threadsPerBlock;

        if (i % 2 == 0) {
            cudaBFSSudoku<<<maxBlocks,threadsPerBlock>>>(old_boards, new_boards, host_count, board_index,params.row,params.col);
            //gpuErrchk(hipMemcpy(&fun, new_boards, N*sizeof(uint64_t), hipMemcpyDeviceToHost));
            params=find_epmty_index(new_boards,params.row,params.col);
        }
        else {
            cudaBFSSudoku<<<maxBlocks,threadsPerBlock>>>(new_boards, old_boards, host_count, board_index,params.row,params.col);
            //gpuErrchk(hipMemcpy(&fun, old_boards, N*sizeof(uint64_t), hipMemcpyDeviceToHost));
            params=find_epmty_index(old_boards,params.row,params.col);
        }
    }

    //gpuErrchk(hipMemcpy(&host_count, board_index, sizeof(int), hipMemcpyDeviceToHost));
    
    // if(zeros % 2 == 0){
    //     gpuErrchk(hipMemcpy(&check, new_boards, N*sizeof(uint64_t), hipMemcpyDeviceToHost));
        print_sudoku_from_b64(new_boards);
    // }
    // else{
    //     gpuErrchk(hipMemcpy(&check, old_boards, N*sizeof(uint64_t), hipMemcpyDeviceToHost));
        print_sudoku_from_b64(old_boards);
    // }
    
    printf("new number of boards retrieved is %d\n", host_count);
    //print_sudoku_from_b64(check);


    gpuErrchk(hipEventRecord(event2));
    gpuErrchk(hipEventSynchronize(event2));
    gpuErrchk(hipDeviceSynchronize());

    hipEventElapsedTime(&dt_ms, event1,event2);
    printf("Time : %f",dt_ms);

    return 0; 
}

