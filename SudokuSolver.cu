#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>  
#include <string.h>
#include <time.h>
#include <math.h>
#include <inttypes.h>

#include "CudaSudoku.cu"
#include "samples.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main(int argc, char* argv[]) {

    hipEvent_t event1,event2;
    
    uint64_t test[N];
    uint64_t check[N];
    uint64_t fun[N];

    uint64_t *new_boards;
    uint64_t *old_boards;
    int *board_index;

    const int sk = pow(2,27);
    int host_count;
    int maxBlocks;
    int zeros;
    params_t params;
    float dt_ms;

    gpuErrchk(hipMallocManaged(&new_boards,sk*sizeof(uint64_t)));
    gpuErrchk(hipMallocManaged(&old_boards,sk*sizeof(uint64_t)));
    gpuErrchk(hipMallocManaged(&board_index,sizeof(int)));

    memset(test,0,N*sizeof(uint64_t));
    memset(check,0,N*sizeof(uint64_t));
    memset(fun,0,N*sizeof(uint64_t));

    gpuErrchk(hipEventCreate(&event1));
    gpuErrchk(hipEventCreate(&event2));

    //------------------------------------------------------------------------------------------------------------------------
    setup_board(new_boards,test9);
    //load("sudoku.txt", test);
    //------------------------------------------------------------------------------------------------------------------------
    
    print_sudoku_from_b64(test);

    zeros=count_zeros(test);
    //gpuErrchk(hipMemcpy(new_boards,test,N*sizeof(uint64_t),hipMemcpyHostToDevice));

    gpuErrchk(hipEventRecord(event1));

    params=find_epmty_index(new_boards,0,0);

    printf("Empty index %i : %i\n",params.row, params.col);

    cudaBFSSudoku<<<1,N>>>(new_boards, old_boards, 1, board_index,params.row,params.col);

    //gpuErrchk(hipMemcpy(&fun, old_boards, N*sizeof(uint64_t), hipMemcpyDeviceToHost))
    params=find_epmty_index(old_boards,params.row,params.col);

    
    for (int i = 0; i<zeros; i++) {

        //gpuErrchk(hipMemcpy(&host_count, board_index, sizeof(int), hipMemcpyDeviceToHost));
        host_cout = board_index;
        printf("total boards after an iteration %d: %d\n", i, host_count);

        gpuErrchk(hipMemset(board_index, 0, sizeof(int)));

        maxBlocks=(N*host_count+256-1)/256;

        if (i % 2 == 0) {
            cudaBFSSudoku<<<maxBlocks,256>>>(old_boards, new_boards, host_count, board_index,params.row,params.col);
            //gpuErrchk(hipMemcpy(&fun, new_boards, N*sizeof(uint64_t), hipMemcpyDeviceToHost));
            params=find_epmty_index(new_boards,params.row,params.col);
        }
        else {
            cudaBFSSudoku<<<maxBlocks,256>>>(new_boards, old_boards, host_count, board_index,params.row,params.col);
            //gpuErrchk(hipMemcpy(&fun, old_boards, N*sizeof(uint64_t), hipMemcpyDeviceToHost));
            params=find_epmty_index(old_boards,params.row,params.col);
        }
    }

    gpuErrchk(hipMemcpy(&host_count, board_index, sizeof(int), hipMemcpyDeviceToHost));
    
    if(zeros % 2 == 0){ // if odd number of iterations run, then send it old boards not new boards;
        //gpuErrchk(hipMemcpy(&check, new_boards, N*sizeof(uint64_t), hipMemcpyDeviceToHost));
        check = new_boards;
    }
    else{
        check = old_boards;
        //gpuErrchk(hipMemcpy(&check, old_boards, N*sizeof(uint64_t), hipMemcpyDeviceToHost));
    }
    
    printf("new number of boards retrieved is %d\n", host_count);
    print_sudoku_from_b64(check);


    gpuErrchk(hipEventRecord(event2));
    gpuErrchk(hipEventSynchronize(event2));
    gpuErrchk(hipDeviceSynchronize());

    hipEventElapsedTime(&dt_ms, event1,event2);
    printf("Time : %f",dt_ms);

    // gpuErrchk(hipFree(new_boards));
    // gpuErrchk(hipFree(old_boards));
    // gpuErrchk(hipFree(board_index));

    return 0; 
}

