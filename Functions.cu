#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 9
#define n 3

typedef struct params{
    int row;
    int col;
}params_t;

__global__ void cudaBFSSudoku(uint64_t *old_boards,
        uint64_t *new_boards,
        int total_boards,
        int *board_index,
        int empty_row,
        int empty_col) {
    
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    unsigned int index=tid/N;

    int attempt = tid - N*index + 1;

    while (index < total_boards) {

        int next_board_index=0;           
        bool works = true;

        int box_row = empty_row/n;
        int box_col = empty_col/n;

        if (!check_row(old_boards+index*N,empty_row,attempt)) 
            works = false;
        else if (!check_col(old_boards+index*N,empty_col,attempt)) 
            works = false;
        else if (!check_box(old_boards+index*N,box_row,box_col,attempt))
            works = false;

        if (works) {
            next_board_index = atomicAdd(board_index, 1);
            for (int i = 0; i < N; i++) {
                new_boards[next_board_index*N+i]=old_boards[index*N+i];
            }
            copy_bits(attempt, &(new_boards+next_board_index*N)[empty_row],0,empty_col*4,4);
            }

        break; 
    }  
}

__device__ __host__ bool findEmptySpot(uint64_t *board, int *row, int *col) {
    for (int r = 0; r < N; r++) {
        for (int c = 0; c < N; c++) {
            if (board[r * N + c] == 0) {
                *row = r;
                *col = c;
                return true;
            }
        }
    }

    return false;
}

__device__ __host__ void print_sudoku(uint64_t *val) {

    for (int i = 0; i < N; i++) {
        if (i % n == 0) {
            printf("------------------------\n");
        }

        for (int j = 0; j < N; j++) {
            if (j % n == 0) {
            printf("| ");
            }
            uint64_t tmp=0;
            copy_bits(val[i],&tmp,j*4,0,4);
            printf("%li ", tmp);
        }

        printf("|\n");
    }
    printf("------------------------\n");
}


__device__ __host__ void setbit(uint64_t val, uint64_t *data, int nshift) { 
	//set bit to data using val as sourse and nshift place where to set
	*data = *data | (val << nshift);
}

__device__ __host__ int getbit(uint64_t input, int nshift) {
	//return value in a certain position 0 or 1
	return (input >> nshift) & 1;
}

__device__ __host__ void copy_bits(uint64_t src, uint64_t *dst, int src_offset, int dst_offset, int len) {
	/*
	src - source of bits
	dst - destanation of bits
	src_offset - starting position from which take bit
	dst_offset - starting position where to set the bit
	len - how many bits
	*/
	for (int i = 0; i < len; i++) {
		setbit(getbit(src, src_offset + i), dst, i + dst_offset);
    }

}

__device__ __host__ void setup_board(uint64_t *src, int *board){
    for(int i=0;i<N;i++){
        for(int j=0;j<N;j++){
            copy_bits(board[i*N+j], &src[i],0,j*4,4);
        }
    }
}


__device__ __host__ params_t find_epmty_index(uint64_t *val, int row, int col){
   
    for(int i=row;i<N;i++){
        for(int j=0;j<N;j++){
            uint64_t tmp=0;
            copy_bits(val[i],&tmp,j*4,0,4);
            if(tmp==0){
                params_t temp={i,j};
                return temp;
            }
        }
    }
    params_t temp={0,0};
    return temp;
}

__device__ __host__ bool check_row(uint64_t *val, int row, int value){
    for(int i=0;i<N;i++){
        uint64_t tmp=0;
        copy_bits(val[row],&tmp,i*4,0,4);
        if(tmp==value)
            return false;
        }
    return true;
}

__device__ __host__ bool check_col(uint64_t *val, int column, int value){
    for(int i=0;i<N;i++){
        uint64_t tmp=0;
        copy_bits(val[i],&tmp,column*4,0,4);
        if(tmp==value)
            return false;
        }
    return true;
}

__device__ __host__ bool check_box(uint64_t *val, int row,int column, int value){
    for(int i=row*n;i<row*n+n;i++){
        for(int j=column*n; j<column*n+n;j++){
            uint64_t tmp=0;
            copy_bits(val[i],&tmp,j*4,0,4);
            if(tmp==value)
                return false;
            }
    }
    return true;
}

__device__ __host__ int count_zeros(uint64_t *val){
    int count=0;
    for(int i=0;i<N;i++){
        for(int j=0; j<N;j++){
            uint64_t tmp=0;
            copy_bits(val[i],&tmp,j*4,0,4);
            if(tmp==0)
                count++;
            }
    }
    return count;
}

